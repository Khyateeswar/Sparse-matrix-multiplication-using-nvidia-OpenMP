#include <bits/stdc++.h>
#include "hip/hip_runtime.h"


#include <omp.h>

using namespace std;

__host__ __device__ void conv(int value,char* ca,int i)
{
    int leftmost_byte;
    int left_middle_byte;
    int right_middle_byte;
    int rightmost_byte;
    
    leftmost_byte = (value & 0x000000FF) >> 0;
    left_middle_byte = (value & 0x0000FF00) >> 8;
    right_middle_byte = (value & 0x00FF0000) >> 16;
    rightmost_byte = (value & 0xFF000000) >> 24;
    char c=*reinterpret_cast<char *>(&leftmost_byte );
    ca[i]=c;
    c=*reinterpret_cast<char *>(&left_middle_byte );
    ca[i+1]=c;
    c=*reinterpret_cast<char *>(&right_middle_byte );
    ca[i+2]=c;
    c=*reinterpret_cast<char *>(&rightmost_byte );
    ca[i+3]=c;
    return ;
}


  __global__ void matmul(int *a, int *b,char* cf,int m,int u,int n,int *p1,int* p2,int *nz,int* fnz) {
  //p1 has the index to access i,j block from a
  //p2 has the index to access i,j block from b
  // nz contains the no of nonzero blocks
  //cf is the char array to send to output

  __shared__ int c[64];// array for storing the block computed by this block
  for(int h=0;h<64;h++){
    c[h]=0;
  }
  __shared__ int p3;
  p3=0;

  __syncthreads();

  int i = blockIdx.x/u;
  int j = blockIdx.x%u;
  int l = threadIdx.x;
  int s = u/(blockDim.x)+1;
 
  for(int k=s*l;k<s*l+s;k++){

    if(i<u && j<u && k<u){
 
    if(p1[i*u+k]>=0 && p2[k*u+j]>=0){
      atomicOr(&p3,1);
      int ik = p1[i*u+k];
      int kj = p2[k*u+j];
      for(int p=0;p<m;p++){
        for(int q=0;q<m;q++){
          for(int z=0;z<m;z++){

            atomicAdd(  &c[p*m+q],b[ kj*m*m+z*m+q]*a[ ik*m*m+p*m+z ]);

          }
        }
      }
    }

    
  }


  }

  __syncthreads();


    if(l==0 && i<u && j<u){
        if( p3>0 ){
            p3 = atomicAdd(nz,1);
            int off = (p3-1)*(4*m*m+8)+12;
            conv(i,cf,off);
            conv(j,cf,off+4);
            int sd=0;
            atomicAdd(fnz,1);
            for(int a=0;a<m;a++){
                for(int b=0;b<m;b++){
                    conv(c[a*m+b],cf,off+8+sd);
                    sd=sd+4;
                }
            }
        }
        if(i==0 && j==0){
          conv(n,cf,0);
          conv(m,cf,4);
          
        }
    }

 
}


__host__  string conv4(int value)
{
    int leftmost_byte;
    int left_middle_byte;
    int right_middle_byte;
    int rightmost_byte;
    
    leftmost_byte = (value & 0x000000FF) >> 0;
    left_middle_byte = (value & 0x0000FF00) >> 8;
    right_middle_byte = (value & 0x00FF0000) >> 16;
    rightmost_byte = (value & 0xFF000000) >> 24;
    string s = "";
    char c=*reinterpret_cast<char *>(&leftmost_byte );
    s=s+c;
    c=*reinterpret_cast<char *>(&left_middle_byte );
    s=s+c;
    c=*reinterpret_cast<char *>(&right_middle_byte );
    s=s+c;
    c=*reinterpret_cast<char *>(&rightmost_byte );
    s=s+c;
    return s;
}

 



int main(int argc, char** argv) {

  if(argc!=4){
        cout<<"Check command line arguments"<<endl;
        return 0;
  }
  char* input_file1 = argv[1];
  char* input_file2 = argv[2];
  char* output_file = argv[3];


  clock_t start,end,starT,endT;
    double timetaken,timeT;

    starT = clock();
    start = clock();


    int n,m,k;

    ifstream ifs;
    ifs.open (input_file1, ios::in | ios::binary );
    ifs.seekg (0, ios::end);
    int length = ifs.tellg();
    ifs.seekg (0, ios::beg);
    char* buf = new char[length];
    ifs.read (buf,length);
    ifs.close();



    n = *reinterpret_cast<int *>( buf );
    m = *reinterpret_cast<int *>( buf+4 );
    k = *reinterpret_cast<int *>( buf+8 );
    int k1=k;
    int u = n/m;

    cout<<n<<" "<<m<<" "<<k<<" "<<endl;

    int* mat1 = new int[k*m*m];
    // #pragma omp parallel for schedule(static)
    for(int i=0;i<k*m*m;i++){
        mat1[i]=0;
    }
    int* p1 = new int[u*u];
    for(int i=0;i<u*u;i++){
        p1[i]=-1;
    }

    
    for(int i=0;i<k;i++){
        int ind = *reinterpret_cast<int *>( buf+(8+m*m*2)*i+12 );
        int jnd = *reinterpret_cast<int *>( buf+(8+m*m*2)*i+4+12);
        p1[ind*u+jnd]=i;
        int ofss = i*m*m;
        int h = (8+m*m*2)*i+8+12;
        for(int j=0;j<m;j++){
            for(int q=0;q<m;q++){
                int val = *reinterpret_cast<uint16_t *>( buf+h);
                mat1[ ofss+j*m+q ] = val;
                h=h+2;
            }
        }
    }
    // cout<<"reading input1 done"<<endl;

    // for(int i=0;i<k*m*m;i++){
    //   cout<<mat1[i]<<" ";
    // }
    // cout<<'\n';


    // for(int i=0;i<u*u;i++){
    //   cout<<p1[i]<<" ";
    // }
    // cout<<'\n';

    ifs.open (input_file2, ios::in | ios::binary );
    ifs.seekg (0, ios::end);
    length = ifs.tellg();
    ifs.seekg (0, ios::beg);
    char* buf1 = new char[length];
    ifs.read (buf1,length);
    ifs.close();

    n = *reinterpret_cast<int *>( buf1 );
    m = *reinterpret_cast<int *>( buf1+4 );
    k = *reinterpret_cast<int *>( buf1+8 );

    cout<<n<<" "<<m<<" "<<k<<" "<<endl;

    int* mat2 = new int[k*m*m];
    // #pragma omp parallel for schedule(static)
    for(int i=0;i<k*m*m;i++){
        mat2[i]=0;
    }

    int* p2 = new int[u*u];
    for(int i=0;i<u*u;i++){
        p2[i]=-1;
    }
    // int p3[u*u];
    // for(int i=0;i<u*u;i++){
    //     p3[i]=0;
    // }
    for(int i=0;i<k;i++){
        int ind = *reinterpret_cast<int *>( buf1+(8+m*m*2)*i+12 );
        int jnd = *reinterpret_cast<int *>( buf1+(8+m*m*2)*i+4+12);
        p2[ind*u+jnd]=i;
        int ofss = i*m*m;
        int h = (8+m*m*2)*i+8+12;
        for(int j=0;j<m;j++){
            for(int q=0;q<m;q++){
                int val = *reinterpret_cast<uint16_t *>( buf1+h);
                mat2[  ofss+j*m+q ] = val;
                h=h+2;;
            }
        }
    }
    // cout<<"reading input2 done"<<endl;

    end = clock();
    timetaken = (end - start) / (double)CLOCKS_PER_SEC;
    cout << "Time taken by taking input: " << fixed << timetaken << "s" << endl;

    // for(int i=0;i<k*m*m;i++){
    //   cout<<mat2[i]<<" ";
    // }
    // cout<<'\n';

    // for(int i=0;i<u*u;i++){
    //   cout<<p2[i]<<" ";
    // }
    // cout<<'\n';

    //int mat3[n*n];



// host copies of variables a, b & c
  int *m1, *m2;
  ///int *m3;
  int *c1, *c2;
  char* ot;
  int *nz;
  int *fnz;
  int fnzz=0;
  int nonzero=1;
// device copies of variables a, b & c

  int size2 = (k*m*m)*sizeof(int);
  int size1 = (k1*m*m)*sizeof(int);
  
  int usize = (u*u)*sizeof(int);
  
  int osize = n*n*2*sizeof(char);
  //cout<<" mem alloc in cpu start "<<endl;
  
  

  start = clock();

// Allocate space for device copies of a, b, c
  hipMalloc((void **)&m1, size1);
  hipMalloc((void **)&m2, size2);
  //cudaMalloc((void **)&m3, (n*n)*sizeof(int));
  hipMalloc((void **)&c1, usize);
  hipMalloc((void **)&c2, usize);
  //cudaMalloc((void **)&c3, usize);
  hipMalloc((void **)&nz, 4);
  hipMalloc((void **)&fnz, 4);
  hipMalloc((void **)&ot, osize);
  

  hipDeviceSynchronize();

  hipMemcpy(nz, &nonzero, 4, hipMemcpyHostToDevice);
  hipMemcpy(fnz, &fnzz, 4, hipMemcpyHostToDevice);
  hipMemcpy(m1, mat1, size1, hipMemcpyHostToDevice);
  hipMemcpy(c1, p1, usize, hipMemcpyHostToDevice);
//cout<<*(d_a+0)<<" "<<*(d_a+1)<<" "<<*(d_a+2)<<endl;
  hipMemcpy(m2, mat2, size2, hipMemcpyHostToDevice);
  hipMemcpy(c2, p2, usize, hipMemcpyHostToDevice);
  //cudaMemcpy(c3, &p3[0], usize, cudaMemcpyHostToDevice);
  
  hipDeviceSynchronize();

  delete [] mat2;
  delete [] mat1;
  delete [] p2;
  delete [] p1;

 
  char* outf= new char[osize];
//cout<<*(d_b+0)<<" "<<*(d_b+1)<<" "<<*(d_b+2)<<endl;
// Launch add() kernel on GPU
  matmul<<<u*u,1000>>>(m1,m2,ot,m,u,n,c1,c2,nz,fnz);
//cout<<*(d_c+0)<<" "<<*(d_c+1)<<" "<<*(d_c+2)<<endl;
// Copy result back to host
  hipDeviceSynchronize();

  hipMemcpy(&nonzero, fnz, 4, hipMemcpyDeviceToHost);
  //cudaDeviceSynchronize();
  // nonzero = nonzero-1;
  //cout<<nonzero<<endl;
  //cudaMemcpy(&mat3[0], m3,(n*n)*sizeof(int) , cudaMemcpyDeviceToHost);
  hipMemcpy(outf, ot,osize , hipMemcpyDeviceToHost);
  //cudaMemcpy(&p3[0], c3, usize, cudaMemcpyDeviceToHost);

  hipDeviceSynchronize();
// Cleanup
  hipFree(m1);
  hipFree(m2);
  hipFree(ot);
  hipFree(nz);
  hipFree(c1);
  hipFree(c2);
  //cudaFree(c3);

  end = clock();
  timetaken = (end - start) / (double)CLOCKS_PER_SEC;
  cout << "Time taken by matrix mul: " << fixed << timetaken << "s" << endl;

  // cout<<" cuda file"<<endl;

  // for(int i=0;i<n;i++){
  //   for(int j=0;j<n;j++){
  //     cout<<mat3[i*n+j]<<" ";
  //   }
  //   cout<<'\n';
  // }

  // for(int i=0;i<u*u;i++){
  //     cout<<p3[i]<<" ";
  //   }
  //   cout<<'\n';



//output the file

start = clock();
cout<<"started writing output"<<endl;

    conv(nonzero,outf,8);
    //cout<<*reinterpret_cast<int *>( outf+8 )<<endl;
    // for(int y=0;y<12+nonzero*(2+m*m)*4;y=y+4){
    //   cout<<*reinterpret_cast<int *>( outf+y )<<endl;
    // }
    ofstream of(output_file,ios::out | ios::binary);
    of.write(outf,12+nonzero*(2+m*m)*4);
    of.close();

    end = clock();
    timetaken = (end - start) / (double)CLOCKS_PER_SEC;
    cout << "Time taken by writing output: " << fixed << timetaken << "s" << endl;

    endT = clock();
    timeT = (endT - starT) / (double)CLOCKS_PER_SEC;
    cout << "Total time taken: " << fixed << timeT << "s" << endl;

  return 0;
}
